#include "hip/hip_runtime.h"
// ----------------------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------------------

/**
 * @file color_app.cu
 *
 * @brief Graph Coloring Gunrock Application
 */

#include <gunrock/gunrock.h>

// Utilities and correctness-checking
#include <gunrock/util/test_utils.cuh>

// Graph definitions
#include <gunrock/app/app_base.cuh>
#include <gunrock/app/test_base.cuh>
#include <gunrock/graphio/graphio.cuh>

// Graph Coloring
#include <gunrock/app/color/color_enactor.cuh>
#include <gunrock/app/color/color_test.cuh>

#include <gunrock/util/info_rapidjson.cuh>

// Others
#include <cstdio>

namespace gunrock {
namespace app {
namespace color {

hipError_t UseParameters(util::Parameters &parameters) {
  hipError_t retval = hipSuccess;
  GUARD_CU(UseParameters_app(parameters));
  GUARD_CU(UseParameters_problem(parameters));
  GUARD_CU(UseParameters_enactor(parameters));
  GUARD_CU(UseParameters_test(parameters));

/*
  GUARD_CU(parameters.Use<unsigned int>(
      "num-colors",
      util::REQUIRED_ARGUMENT | util::SINGLE_VALUE | util::INTERNAL_PARAMETER,
      0, "number of output colors", __FILE__, __LINE__));
*/

  GUARD_CU(parameters.Use<std::string>(
      "tag", util::REQUIRED_ARGUMENT | util::OPTIONAL_PARAMETER, "",
      "tag info for json string", __FILE__, __LINE__));

  GUARD_CU(parameters.Use<bool>(
      "check-percentage", util::REQUIRED_ARGUMENT | util::OPTIONAL_PARAMETER, false,
      "Display the percentage of number of nodes that are colored (default=false)",
      __FILE__, __LINE__));

  GUARD_CU(parameters.Use<bool>(
      "loop-color", util::REQUIRED_ARGUMENT | util::OPTIONAL_PARAMETER, true,
      "Serially compare rand to all node neighbor, set to false to use advance \
      neighbor reduce (default=true)",
      __FILE__, __LINE__));

  GUARD_CU(parameters.Use<bool>(
      "min-color", util::REQUIRED_ARGUMENT | util::OPTIONAL_PARAMETER, true,
      "Enable coloring with minimum independent set as well as \
      maximum(default=true)",
      __FILE__, __LINE__));

  GUARD_CU(parameters.Use<bool>(
      "test-run", util::REQUIRED_ARGUMENT | util::OPTIONAL_PARAMETER, false,
      "Perform test run to atomically generate max iteration (default=true)",
      __FILE__, __LINE__));

  GUARD_CU(parameters.Use<int>(
      "user-iter",
      util::REQUIRED_ARGUMENT | util::SINGLE_VALUE | util::OPTIONAL_PARAMETER,
      3, "Number of iterations color should run for (default=3).", __FILE__,
      __LINE__));

  GUARD_CU(parameters.Use<bool>(
      "JPL", util::REQUIRED_ARGUMENT | util::OPTIONAL_PARAMETER, false,
      "Use JPL exact coloring method (true=use JPL).", __FILE__, __LINE__));

  GUARD_CU(parameters.Use<int>(
      "no-conflict", util::REQUIRED_ARGUMENT | util::OPTIONAL_PARAMETER, 0,
      "Resolve color conflict, 0 to skip check, 1 to check at end of\
      every iteration with random,\
      2 to check at end of every iteration with degree(default = 0).",
      __FILE__, __LINE__));

  GUARD_CU(parameters.Use<int>(
      "prohibit-size", util::REQUIRED_ARGUMENT | util::OPTIONAL_PARAMETER, 0,
      "Needed to allocate memory for hash function, if parameter is\
      positive,\
      hash coloring is used instead of random coloring (default = 0).",
      __FILE__, __LINE__));

  GUARD_CU(parameters.Use<int>(
      "seed", util::REQUIRED_ARGUMENT | util::OPTIONAL_PARAMETER, time(NULL),
      "seed for random number generator", __FILE__, __LINE__));

  GUARD_CU(parameters.Use<bool>(
      "LBCOLOR", util::REQUIRED_ARGUMENT | util::OPTIONAL_PARAMETER, false,
      "load balancing enabled for graph coloring (true=neighbor_reduce)",
      __FILE__, __LINE__));

  return retval;
}

/**
 * @brief Run color tests
 * @tparam     GraphT        Type of the graph
 * @tparam     ValueT        Type of the distances
 * @param[in]  parameters    Excution parameters
 * @param[in]  graph         Input graph
...
 * @param[in]  target        where to perform the app
 * \return hipError_t error message(s), if any
 */
template <typename GraphT>
hipError_t RunTests(util::Parameters &parameters, GraphT &graph,
                     bool color_balance, typename GraphT::VertexT *ref_colors,
                     util::Location target) {
  hipError_t retval = hipSuccess;

  typedef typename GraphT::VertexT VertexT;
  typedef typename GraphT::ValueT ValueT;
  typedef typename GraphT::SizeT SizeT;
  typedef Problem<GraphT> ProblemT;
  typedef Enactor<ProblemT> EnactorT;

  // CLI parameters
  bool quiet_mode = parameters.Get<bool>("quiet");
  int num_runs = parameters.Get<int>("num-runs");
  std::string validation = parameters.Get<std::string>("validation");
  util::Info info("color", parameters, graph);

  util::CpuTimer cpu_timer, total_timer;
  cpu_timer.Start();
  total_timer.Start();

  VertexT *h_colors = new VertexT[graph.nodes];

  // Allocate problem and enactor on GPU, and initialize them
  ProblemT problem(parameters);
  EnactorT enactor;
  GUARD_CU(problem.Init(graph, target));
  GUARD_CU(enactor.Init(problem, target));

  cpu_timer.Stop();
  parameters.Set("preprocess-time", cpu_timer.ElapsedMillis());
  int num_colors = 0;
  for (int run_num = 0; run_num < num_runs; ++run_num) {
    GUARD_CU(problem.Reset(target));
    GUARD_CU(enactor.Reset(target));

    util::PrintMsg("__________________________", !quiet_mode);

    cpu_timer.Start();
    GUARD_CU(enactor.Enact());
    cpu_timer.Stop();
    info.CollectSingleRun(cpu_timer.ElapsedMillis());

    util::PrintMsg(
        "--------------------------\nRun " + std::to_string(run_num) +
            " elapsed: " + std::to_string(cpu_timer.ElapsedMillis()) +
            ", #iterations = " +
            std::to_string(enactor.enactor_slices[0].enactor_stats.iteration),
        !quiet_mode);
    if (validation == "each") {
      GUARD_CU(problem.Extract(h_colors));
      SizeT num_errors = Validate_Results(parameters, graph, h_colors,
                                          ref_colors, &num_colors, false);
    }
  }

  cpu_timer.Start();

  GUARD_CU(problem.Extract(h_colors));
  if (validation == "last") {
    SizeT num_errors = Validate_Results(parameters, graph, h_colors, ref_colors,
                                        &num_colors, false);
  }
  printf("Number of colors needed: %d\n", num_colors);

  UseParameters_test(parameters);
  // parameters.Set("num-colors", num_colors);
  info.SetVal("num-colors", std::to_string(num_colors));

  // compute running statistics
  // h_distances
  info.ComputeTraversalStats(enactor, (VertexT *)NULL);
// Display_Memory_Usage(problem);
#ifdef ENABLE_PERFORMANCE_PROFILING
  // Display_Performance_Profiling(enactor);
#endif

  // Clean up
  GUARD_CU(enactor.Release(target));
  GUARD_CU(problem.Release(target));
  delete[] h_colors;
  h_colors = NULL;
  cpu_timer.Stop();
  total_timer.Stop();

  info.Finalize(cpu_timer.ElapsedMillis(), total_timer.ElapsedMillis());
  return retval;
}

}  // namespace color
}  // namespace app
}  // namespace gunrock

// ===========================================================================================
// ========================= CODE BELOW THIS LINE NOT NEEDED FOR TESTS
// =======================
// ===========================================================================================

// /*
// * @brief Entry of gunrock_template function
// * @tparam     GraphT     Type of the graph
// * @tparam     ValueT     Type of the distances
// * @param[in]  parameters Excution parameters
// * @param[in]  graph      Input graph
// * @param[out] distances  Return shortest distance to source per vertex
// * @param[out] preds      Return predecessors of each vertex
// * \return     double     Return accumulated elapsed times for all runs
// */
// template <typename GraphT, typename ValueT = typename GraphT::ValueT>
// double gunrock_Template(
//     gunrock::util::Parameters &parameters,
//     GraphT &graph
//     //ValueT **distances
//     )
// {
//     typedef typename GraphT::VertexT VertexT;
//     typedef gunrock::app::Template::Problem<GraphT  > ProblemT;
//     typedef gunrock::app::Template::Enactor<ProblemT> EnactorT;
//     gunrock::util::CpuTimer cpu_timer;
//     gunrock::util::Location target = gunrock::util::DEVICE;
//     double total_time = 0;
//     if (parameters.UseDefault("quiet"))
//         parameters.Set("quiet", true);

//     // Allocate problem and enactor on GPU, and initialize them
//     ProblemT problem(parameters);
//     EnactorT enactor;
//     problem.Init(graph  , target);
//     enactor.Init(problem, target);

//     int num_runs = parameters.Get<int>("num-runs");
//     // std::vector<VertexT> srcs =
//     parameters.Get<std::vector<VertexT>>("srcs");
//     // int num_srcs = srcs.size();
//     for (int run_num = 0; run_num < num_runs; ++run_num)
//     {
//         // int src_num = run_num % num_srcs;
//         // VertexT src = srcs[src_num];
//         problem.Reset(/*src,*/ target);
//         enactor.Reset(/*src,*/ target);

//         cpu_timer.Start();
//         enactor.Enact(/*src*/);
//         cpu_timer.Stop();

//         total_time += cpu_timer.ElapsedMillis();
//         problem.Extract(/*distances[src_num]*/);
//     }

//     enactor.Release(target);
//     problem.Release(target);
//     // srcs.clear();
//     return total_time;
// }

//  * @brief Simple interface take in graph as CSR format
//  * @param[in]  num_nodes   Number of veritces in the input graph
//  * @param[in]  num_edges   Number of edges in the input graph
//  * @param[in]  row_offsets CSR-formatted graph input row offsets
//  * @param[in]  col_indices CSR-formatted graph input column indices
//  * @param[in]  edge_values CSR-formatted graph input edge weights
//  * @param[in]  num_runs    Number of runs to perform SSSP
//  * @param[in]  sources     Sources to begin traverse, one for each run
//  * @param[in]  mark_preds  Whether to output predecessor info
//  * @param[out] distances   Return shortest distance to source per vertex
//  * @param[out] preds       Return predecessors of each vertex
//  * \return     double      Return accumulated elapsed times for all runs

// template <
//     typename VertexT = int,
//     typename SizeT   = int,
//     typename GValueT = unsigned int,
//     typename TValueT = GValueT>
// float Template(
//     const SizeT        num_nodes,
//     const SizeT        num_edges,
//     const SizeT       *row_offsets,
//     const VertexT     *col_indices,
//     const GValueT     *edge_values,
//     const int          num_runs
//     //      VertexT     *sources,
//     //      SSSPValueT **distances
//     )
// {
//     typedef typename gunrock::app::TestGraph<VertexT, SizeT, GValueT,
//         gunrock::graph::HAS_EDGE_VALUES | gunrock::graph::HAS_CSR>
//         GraphT;
//     typedef typename GraphT::CsrT CsrT;

//     // Setup parameters
//     gunrock::util::Parameters parameters("Template");
//     gunrock::graphio::UseParameters(parameters);
//     gunrock::app::Template::UseParameters(parameters);
//     gunrock::app::UseParameters_test(parameters);
//     parameters.Parse_CommandLine(0, NULL);
//     parameters.Set("graph-type", "by-pass");
//     parameters.Set("num-runs", num_runs);
//     // std::vector<VertexT> srcs;
//     // for (int i = 0; i < num_runs; i ++)
//     //     srcs.push_back(sources[i]);
//     // parameters.Set("srcs", srcs);

//     bool quiet = parameters.Get<bool>("quiet");
//     GraphT graph;
//     // Assign pointers into gunrock graph format
//     graph.CsrT::Allocate(num_nodes, num_edges, gunrock::util::HOST);
//     graph.CsrT::row_offsets   .SetPointer(row_offsets, gunrock::util::HOST);
//     graph.CsrT::column_indices.SetPointer(col_indices, gunrock::util::HOST);
//     graph.CsrT::edge_values   .SetPointer(edge_values, gunrock::util::HOST);
//     graph.FromCsr(graph.csr(), true, quiet);
//     gunrock::graphio::LoadGraph(parameters, graph);

//     // Run the Template
//     double elapsed_time = gunrock_Template(parameters, graph /*,
//     distances*/);

//     // Cleanup
//     graph.Release();
//     // srcs.clear();

//     return elapsed_time;
// }

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:
